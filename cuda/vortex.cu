#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <getopt.h>
#include <errno.h>
#include <fcntl.h>
#include <math.h>
#include <time.h>

#include "data.h"
#include "vtk.h"
#include "setup.h"
#include "kernels.cuh"
#include "args.h"

struct timespec timer;


double get_time()
{
    clock_gettime(CLOCK_MONOTONIC, &timer);
    return (double)(timer.tv_sec + timer.tv_nsec / 1000000000.0);
}

void boundary_conditions(dim3 threads, dim3 blocks) {
    // Creates streams to run kernels in parallel
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    // Runs kernels WE and NS in parallel
    boundary_conditions_WE_kernel<<<blocks, threads, 0, s1>>>(u, v);
    boundary_conditions_NS_kernel<<<blocks, threads, 0, s2>>>(u, v);

    // Ensures last two kernels have completed
    hipStreamSynchronize(s1);
    hipStreamSynchronize(s2);

    // Runs noslip and boundary condition kernels in parallel
    boundary_conditions_noslip_kernel<<<blocks, threads, 0, s1>>>(u, v, flag);
    apply_boundary_conditions_west_edge_kernel<<<blocks, threads, 0, s2>>>(u, v);

    // Closes additional streams when completed
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);

    // Makes function inline with default stream
    hipDeviceSynchronize();
}

void timestep_interval(dim3 threads, dim3 blocks, int reduction_threads){
    // Creates streams to run kernels in parallel
    abs_max_reduction_blocks_kernel<<<blocks, threads, threads.x * threads.y * sizeof(double)>>>(u, umax_red, 0);
    abs_max_reduction_blocks_kernel<<<blocks, threads, threads.x * threads.y * sizeof(double)>>>(v, vmax_red, 1);
    abs_max_reduction_global_kernel<<<1, reduction_threads, reduction_threads * sizeof(double)>>>(umax_red, umax_g, blocks.x, blocks.y);
    abs_max_reduction_global_kernel<<<1, reduction_threads, reduction_threads * sizeof(double)>>>(vmax_red, vmax_g, blocks.x, blocks.y);

    // Completes the final sequential part of 
    set_timestep_interval_kernel<<<1, 1>>>(umax_g, vmax_g);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&del_t_h, HIP_SYMBOL(del_t), sizeof(double));
}

void compute_tentative_velocity(dim3 threads, dim3 blocks){
    hipStream_t s1, s2, s3, s4;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);
    hipStreamCreate(&s3);
    hipStreamCreate(&s4);
    
    tentative_velocity_update_f_kernel<<<blocks, threads, 0, s1>>>(u, v, f, flag);
    tentative_velocity_update_g_kernel<<<blocks, threads, 0, s2>>>(u, v, g, flag);
    tentative_velocity_g_boundaries_kernel<<<blocks, threads, 0, s3>>>(g, v);
    tentative_velocity_f_boundaries_kernel<<<blocks, threads, 0, s4>>>(f, u);
    hipDeviceSynchronize();

    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipStreamDestroy(s3);
    hipStreamDestroy(s4);
    
    // Makes function inline with default stream
    hipDeviceSynchronize();
    }

void compute_rhs(dim3 threads, dim3 blocks) {
    compute_rhs_kernel<<<blocks, threads>>>(u, v, p, rhs, f, g, flag);
}

void poisson(dim3 threads, dim3 blocks, int reduction_threads)
{
    /* p0 Reduction*/
    p0_reduction_blocks_kernel<<<blocks, threads, threads.x * threads.y * sizeof(double)>>>(p, flag, p0_reductions);
    p0_reduction_global_kernel<<<1, threads, reduction_threads * sizeof(double)>>>(p0_reductions, p0, blocks.x, blocks.y);

    /* Red/Black SOR-iteration */
    for (int iter = 0; iter < itermax; iter++)
    {
        // Star computation for even indicies then odd indicies
        star_computation_kernel<<<blocks, threads>>>(u, v, p, rhs, f, g, flag, 0);
        star_computation_kernel<<<blocks, threads>>>(u, v, p, rhs, f, g, flag, 1);

        /* Residual Reduction */
        residual_reduction_blocks_kernel<<<blocks, threads, threads.x * threads.y * sizeof(double)>>>(p, rhs, flag, residual_reductions);
        residual_reduction_global_kernel<<<1, reduction_threads, reduction_threads * sizeof(double)>>>(residual_reductions, residual, blocks.x, blocks.y, p0);
        hipDeviceSynchronize();

        // Copies residual to host code so it can be checked against eps (and printed in main vortex loop)
        hipMemcpy(&residual_h, residual, sizeof(double), hipMemcpyDeviceToHost);

        /* convergence? */
        if (residual_h < eps)
            break;
    }
}

void update_velocity(dim3 threads, dim3 blocks) {
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    update_velocity_u_kernel<<<blocks, threads, 0, s1>>>(u, v, p, rhs, f, g, flag);
    update_velocity_v_kernel<<<blocks, threads, 0, s2>>>(u, v, p, rhs, f, g, flag);
    hipDeviceSynchronize();

    hipStreamDestroy(s1);
    hipStreamDestroy(s2);

    // Makes function inline with default stream
    hipDeviceSynchronize();
}

void program_start(dim3 threads, dim3 blocks, int argc, char *argv[]){
    set_defaults();
    parse_args(argc, argv);

    setup();
    hipDeviceSynchronize();

    if (verbose)
        print_opts();

    allocate_arrays();

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    
    setup_uvp_kernel<<<blocks, threads, 0, stream1>>>(u, v, p);
    setup_flag_kernel<<<blocks, threads, 0, stream2>>>(flag);
    hipDeviceSynchronize();

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    // Makes function inline with default stream
    hipDeviceSynchronize();

    boundary_conditions(threads, blocks);
}


/**
 * @brief The main routine that sets up the problem and executes the solving routines routines
 *
 * @param argc The number of arguments passed to the program
 * @param argv An array of the arguments passed to the program
 * @return int The return value of the application
 */
int main(int argc, char *argv[])
{
    // Number of threads and blocks required when running one thread per grid cell
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((imax_h + 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
				   (jmax_h + 2 + threadsPerBlock.y - 1) / threadsPerBlock.y);
    int reduction_threads = pow(2, ceil(log2(numBlocks.x * numBlocks.y)));

    /* Timer Initialisations */
    double total_time = get_time();

    program_start(threadsPerBlock, numBlocks, argc, argv);

    /* Main loop */
    int iters = 0;
    double t;
    for (t = 0.0; t < t_end_h; t += del_t_h, iters++)
    {
        if (!fixed_dt) {
            timestep_interval(threadsPerBlock, numBlocks, reduction_threads);
        }

        compute_tentative_velocity(threadsPerBlock, numBlocks);

        compute_rhs(threadsPerBlock, numBlocks);

        poisson(threadsPerBlock, numBlocks, reduction_threads);

        update_velocity(threadsPerBlock, numBlocks);

        boundary_conditions(threadsPerBlock, numBlocks);

        if ((iters % output_freq == 0))
        {
            printf("Step %8d, Time: %14.8e (del_t: %14.8e), Residual: %14.8e\n", iters, t + del_t_h, del_t_h, residual_h);

            if ((!no_output) && (enable_checkpoints)) {
                hipDeviceSynchronize();
                update_host_arrays();
                write_checkpoint(iters, t + del_t_h);
            }
        }
    } /* End of main loop */
 
    hipDeviceSynchronize();
    update_host_arrays();

    total_time = get_time() - total_time;

    printf("Step %8d, Time: %14.8e, Residual: %14.8e\n", iters, t, residual_h);
    printf("Simulation complete.\n");

    fprintf(stderr, "Timing Summary\n");
    fprintf(stderr, "Total Time: %lf\n", total_time);

    if (!no_output)
        write_result(iters, t);

    free_arrays();

    return 0;
}