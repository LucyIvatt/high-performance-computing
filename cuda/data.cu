#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>

#include "vtk.h"
#include "data.h"

double xlength = 4.0; /* Width of simulated domain */
double ylength = 1.0; /* Height of simulated domain */
int imax = 512;		  /* Number of cells horizontally */
int jmax = 128;		  /* Number of cells vertically */

double t_end = 5.0;	  /* Simulation runtime */
double del_t = 0.003; /* Duration of each timestep */
double tau = 0.5;	  /* Safety factor for timestep control */

int itermax = 100;	/* Maximum number of iterations in SOR */
double eps = 0.001; /* Stopping error threshold for SOR */
double omega = 1.7; /* Relaxation parameter for SOR */
double y = 0.9;		/* Gamma, Upwind differencing factor in PDE discretisation */

double Re = 500.0; /* Reynolds number */
double ui = 1.0;   /* Initial X velocity */
double vi = 0.0;   /* Initial Y velocity */

double delx, dely;

int fluid_cells = 0;

// Grids used for veclocities, pressure, rhs, flag and temporary f and g arrays
int u_size_x, u_size_y;
double **u;
int v_size_x, v_size_y;
double **v;
int p_size_x, p_size_y;
double **p;
int rhs_size_x, rhs_size_y;
double **rhs;
int f_size_x, f_size_y;
double **f;
int g_size_x, g_size_y;
double **g;
int flag_size_x, flag_size_y;
char **flag;

/**
 * @brief Allocate a 2D array that is addressable using square brackets
 *
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @return double** A 2D array
 */
double **alloc_2d_array(int m, int n)
{
	double **x;
	int i;

	x = (double **)malloc(m * sizeof(double *));
	x[0] = (double *)calloc(m * n, sizeof(double));
	for (i = 1; i < m; i++)
		x[i] = &x[0][i * n];
	return x;
}

/**
 * @brief Allocate a 2D char array that is addressable using square brackets
 *
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @return char** A 2D array
 */
char **alloc_2d_char_array(int m, int n)
{
	char **x;
	int i;

	x = (char **)malloc(m * sizeof(char *));
	x[0] = (char *)calloc(m * n, sizeof(char));
	for (i = 1; i < m; i++)
		x[i] = &x[0][i * n];
	return x;
}

/**
 * @brief Allocate a 2D array that is addressable using square brackets
 *
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @return double** A 2D array
 */
double **alloc_2d_array_cuda(int m, int n)
{
	double** x;

	hipMalloc(&x, m * sizeof(double *));
	hipMemset(&(x[0]), 0, m * n * sizeof(double));
	
	for (int i = 1; i < m; i++)
		x[i] = &x[0][i * n];

	return x;
}

/**
 * @brief Allocate a 2D char array that is addressable using square brackets
 *
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @return char** A 2D array
 */
char **copy_char_array_to_device(int m, int n, char **src)
{
	char **dest;

	hipMalloc((void **) &dest, m * sizeof(char *));
	hipMalloc((void **) &dest[0], m * n * sizeof(char));

	hipMemcpy(dest[0], src[0], m * n * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(dest, src, m * sizeof(char *), hipMemcpyHostToDevice);
	return dest;
}

/**
 * @brief Allocate a 2D char array that is addressable using square brackets
 *
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @return char** A 2D array
 */
double **copy_double_array_to_device(int m, int n, double **src)
{
	double **dest;

	hipMalloc((void **) &dest, m * sizeof(double *));
	hipMalloc((void **) &dest[0], m * n * sizeof(double));

	hipMemcpy(dest[0], src[0], m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dest, src, m * sizeof(double *), hipMemcpyHostToDevice);
	return dest;
}

/**
 * @brief Free a 2D array
 *
 * @param array The 2D array to free
 */
void free_2d_array_device(void **array)
{
	hipFree(array[0]);
	hipFree(array);
}

/**
 * @brief Free a 2D array
 *
 * @param array The 2D array to free
 */
void free_2d_array_host(void **array)
{
	free(array[0]);
	free(array);
}

