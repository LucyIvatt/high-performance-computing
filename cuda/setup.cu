#include <stdio.h>
#include <stdlib.h>

#include "data.h"
#include "vtk.h"
#include "kernels.cuh"

/**
 * @brief Set up some default values before arguments are parsed.
 *
 */
void set_defaults()
{
    set_default_base();
}

/**
 * @brief Set up some values after arguments have been parsed, copies to GPU constants as needed.
 *
 */
void setup()
{
    // Values taken from arg inputs
    hipMemcpyToSymbol(HIP_SYMBOL(imax), &imax_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(jmax), &jmax_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(t_end), &t_end_h, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(del_t), &del_t_h, sizeof(double));

    // Values calculated from arg inputs
    delx_h = xlength / imax_h;
    dely_h = ylength / jmax_h;

    hipMemcpyToSymbol(HIP_SYMBOL(delx), &delx_h, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dely), &dely_h, sizeof(double));

    double rdx2_h = 1.0 / (delx_h * delx_h);
    double rdy2_h = 1.0 / (dely_h * dely_h);
    double omega_h = 1.7;
    double beta_2_h = -omega_h / (2.0 * (rdx2_h + rdy2_h));

    hipMemcpyToSymbol(HIP_SYMBOL(rdx2), &rdx2_h, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(rdy2), &rdy2_h, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(beta_2), &beta_2_h, sizeof(double));

    double mx_h = 20.0 / 41.0 * jmax_h * dely_h;
    double rad1_h = 5.0 / 41.0 * jmax_h * dely_h;

    hipMemcpyToSymbol(HIP_SYMBOL(mx), &mx_h, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(my), &mx_h, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(rad1), &rad1_h, sizeof(double));

    int fluid_cells_h = imax_h * jmax_h;
    hipMemcpyToSymbol(HIP_SYMBOL(fluid_cells), &fluid_cells_h, sizeof(int));
}

/**
 * @brief Allocate all of the arrays used by the computation.
 *
 */
void allocate_arrays()
{
    /* Allocate arrays */
    u_size_x_h = imax_h + 2;
    u_size_y_h = jmax_h + 2;
    u_host = alloc_2d_array(u_size_x_h, u_size_y_h);
    u = copy_2d_array_to_gpu(u_host, u_size_x_h, u_size_y_h);
    hipMemcpyToSymbol(HIP_SYMBOL(u_size_x), &u_size_x_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(u_size_y), &u_size_y_h, sizeof(int));

    v_size_x_h = imax_h + 2;
    v_size_y_h = jmax_h + 2;
    v_host = alloc_2d_array(v_size_x_h, v_size_y_h);
    v = copy_2d_array_to_gpu(v_host, v_size_x_h, v_size_y_h);
    hipMemcpyToSymbol(HIP_SYMBOL(v_size_x), &v_size_x_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(v_size_y), &v_size_y_h, sizeof(int));


    f_size_x_h = imax_h + 2;
    f_size_y_h = jmax_h + 2;
    f_host = alloc_2d_array(f_size_x_h, f_size_y_h);
    f = copy_2d_array_to_gpu(f_host, f_size_x_h, f_size_y_h);
    hipMemcpyToSymbol(HIP_SYMBOL(f_size_x), &f_size_x_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(f_size_y), &f_size_y_h, sizeof(int));


    g_size_x_h = imax_h + 2;
    g_size_y_h = jmax_h + 2;
    g_host = alloc_2d_array(g_size_x_h, g_size_y_h);
    g = copy_2d_array_to_gpu(g_host, g_size_x_h, g_size_y_h);
    hipMemcpyToSymbol(HIP_SYMBOL(g_size_x), &g_size_x_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(g_size_y), &g_size_y_h, sizeof(int));

    p_size_x_h = imax_h + 2;
    p_size_y_h = jmax_h + 2;
    p_host = alloc_2d_array(p_size_x_h, p_size_y_h);
    p = copy_2d_array_to_gpu(p_host, p_size_x_h, p_size_y_h);
    hipMemcpyToSymbol(HIP_SYMBOL(p_size_x), &p_size_x_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(p_size_y), &p_size_y_h, sizeof(int));

    rhs_size_x_h = imax_h + 2;
    rhs_size_y_h = jmax_h + 2;
    rhs_host = alloc_2d_array(rhs_size_x_h, rhs_size_y_h);
    rhs = copy_2d_array_to_gpu(rhs_host, rhs_size_x_h, rhs_size_y_h);
    hipMemcpyToSymbol(HIP_SYMBOL(rhs_size_x), &rhs_size_x_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(rhs_size_y), &rhs_size_y_h, sizeof(int));

    flag_size_x_h = imax_h + 2;
    flag_size_y_h = jmax_h + 2;
    flag_host = alloc_2d_char_array(flag_size_x_h, flag_size_y_h);
    flag = copy_2d_char_array_to_gpu(flag_host, flag_size_x_h, flag_size_y_h);
    hipMemcpyToSymbol(HIP_SYMBOL(flag_size_x), &flag_size_x_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(flag_size_y), &flag_size_y_h, sizeof(int));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((imax_h + 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
				   (jmax_h + 2 + threadsPerBlock.y - 1) / threadsPerBlock.y);

    p0 = allocate_2d_gpu_array(1, 1);
    p0_reductions = allocate_2d_gpu_array(numBlocks.x, numBlocks.y);

    residual = allocate_2d_gpu_array(1, 1);
    residual_reductions = allocate_2d_gpu_array(numBlocks.x, numBlocks.y);

    umax_g = allocate_2d_gpu_array(1, 1);
    vmax_g = allocate_2d_gpu_array(1, 1);

    umax_red = allocate_2d_gpu_array(numBlocks.x, numBlocks.y);
    vmax_red = allocate_2d_gpu_array(numBlocks.x, numBlocks.y);


    if (!u_host || !v_host || !f_host || !g_host || !p_host || !rhs_host || !flag_host)
    {
        fprintf(stderr, "Couldn't allocate memory for matrices.\n");
        exit(1);
    }
}

void update_host_arrays() {
    update_host_array(u_host, u, u_size_x_h, u_size_y_h);
    update_host_array(v_host, v, v_size_x_h, v_size_y_h);
    update_host_array(f_host, f, f_size_x_h, f_size_y_h);
    update_host_array(g_host, g, g_size_x_h, g_size_y_h);
    update_host_array(p_host, p, p_size_x_h, p_size_y_h);
    update_host_array(rhs_host, rhs, rhs_size_x_h, rhs_size_y_h);
    update_host_char_array(flag_host, flag, flag_size_x_h, flag_size_y_h);
}

/**
 * @brief Free all of the arrays used for the computation.
 *
 */
void free_arrays()
{
    free_2d_array((void *)u_host);
    free_2d_array((void *)v_host);
    free_2d_array((void *)f_host);
    free_2d_array((void *)g_host);
    free_2d_array((void *)p_host);
    free_2d_array((void *)rhs_host);
    free_2d_array((void *)flag_host);

    free_gpu_array((void *)u);
    free_gpu_array((void *)v);
    free_gpu_array((void *)f);
    free_gpu_array((void *)g);
    free_gpu_array((void *)p);
    free_gpu_array((void *)rhs);
    free_gpu_array((void *)flag);
}

